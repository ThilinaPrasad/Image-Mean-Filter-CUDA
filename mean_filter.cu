#include "hip/hip_runtime.h"
// Compile file => nvcc <input_filename.cu> -o image3 <output_filename>
// Run file => ./<filename> <bmp image> <window size>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

unsigned char* mean_filter_h(unsigned char* matrix, int width, int height, int window_size) {
	int ignored_length = window_size/2;
	for(int i = ignored_length; i < (height-ignored_length); i++) {
		for(int j = ignored_length; j < (width-ignored_length); j++) {			
			// filter window handling
			int val_sum = 0;
			for(int k=i-ignored_length; k<=i+ignored_length;k++){
				for(int l=j-ignored_length;l<=j+ignored_length;l++){
					val_sum += matrix[(k)*width + (l)];
					// printf("%d ",matrix[(k)*width + (l)]);
				}
				// printf("\n");
			}
			// printf("===============================================\n");
			matrix[i*width + j] = val_sum/(window_size*window_size);
		}
	}
	return matrix;
}

__global__ void mean_filter_d(unsigned char* matrix, int length, int window_size) {
	int ignored = window_size/2;
	int i = blockIdx.x + ignored;
	int j = threadIdx.x + ignored;
	
	int val_sum = 0;
	for(int k = i-ignored; k < i+ignored+1; k++){
		for(int l = j-ignored; l < j+ignored+1; l++){
			val_sum += matrix[k*length + l];
		}
	}
	matrix[i*length + j] = val_sum/(window_size*window_size);
}

void saveBmp(unsigned char* info,unsigned char** rows,int row_padded,int height,char* filename){
    // write to output_file
    char outputfile[15];
    sprintf(outputfile,filename,height);

	FILE* fw = fopen(outputfile, "wb");
	fwrite(info, 54*sizeof(unsigned char), 1, fw);
	for(int i = 0; i<height; i++) {
		fwrite(rows[i], row_padded*sizeof(unsigned char), 1, fw);
	}
	fclose(fw);
}

int main(int argc, const char * argv[]) {
	
	const char* input_file = argv[1];
	int window_size = strtol(argv[2], NULL, 10);
	
	FILE* fr = fopen(input_file, "rb");
	if(fr == NULL) {
		return 2;
	}

	// read image header (54 bytes) and capture height and wigth
    unsigned char* info = (unsigned char*)malloc(54*sizeof(unsigned char));
    fread(info, sizeof(unsigned char), 54, fr); // read the 54-byte header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];

	printf("Input file: %s \n",input_file);
	printf("Window size: %d \n",window_size);
	printf("Dimensions(%d,%d): \n", width, height);

	// row padding fix
    int row_padded = (width*3 + 3) & (~3);
	
	// pixel array (row major)
	unsigned char* matrix = (unsigned char*)malloc(height*width*sizeof(unsigned char));
	
	// binary data row matrix
	unsigned char** rows = (unsigned char**) malloc(height*sizeof(unsigned char*));
	unsigned char tmp;

	// fill pixel matrix and binary data matrix
    for(int i = 0; i < height; i++) {
		unsigned char* data = (unsigned char*)malloc(row_padded*sizeof(unsigned char));
        fread(data, sizeof(unsigned char), row_padded, fr);
        for(int j = 0; j < width*3; j += 3) {
            // Convert (B, G, R) to (R, G, B)
            tmp = data[j];
            data[j] = data[j+2];
            data[j+2] = tmp;
			//printf("i - %d, j - %d >> R-%d, G-%d, B-%d\n", i, j, (int)data[j], (int)data[j+1], (int)data[j+2]);
			int im = height-i-1; 
			int jm = j/3;
			matrix[im*width + jm] = data[j];
        }
		rows[i] = data;
    }
    
    // get cpu output to matrix
	unsigned char* matrix_out_h = (unsigned char*)malloc(height*width*sizeof(unsigned char));

	//CPU pixel array operation
   	printf("Running CPU Mean filter... ");
    clock_t start_h = clock();
	matrix_out_h = mean_filter_h(matrix,width,height,window_size);
    clock_t end_h = clock();
	printf(" => Done \n");

    printf("Saving CPU output... ");
	// save changes in binary data matrix from CPU
	for(int i = 0; i < height; i++) {
		for(int j = 0; j < width*3; j += 3) {
			int im = height-i-1; 
			int jm = j/3;
            rows[i][j] = matrix_out_h[im*width + jm];
			rows[i][j+1] = matrix_out_h[im*width + jm];
			rows[i][j+2] = matrix_out_h[im*width + jm];
        }
    }
	printf(" => Done \n");

	// Write CPU output
    saveBmp(info,rows,row_padded,height,"CPU_out_%d.bmp");

    // GPU pixel array operation
	int ignored = window_size/2;
	int grid_size = height - 2*ignored;
	int block_size = width - 2*ignored;
	
	unsigned char* matrix_d;
    // get GPU output to matrix
	unsigned char* matrix_out_d = (unsigned char*)malloc(height*width*sizeof(unsigned char));
	
	hipMalloc((void **)&matrix_d, height*width*sizeof(unsigned char));
	hipMemcpy(matrix_d, matrix, height*width*sizeof(unsigned char), hipMemcpyHostToDevice);
	printf("Running GPU Mean filter... ");
    clock_t start_d=clock();
	mean_filter_d<<<grid_size, block_size>>>(matrix_d, width, window_size);
    hipDeviceSynchronize();
    clock_t end_d=clock();
	printf(" => Done \n");
	hipMemcpy(matrix_out_d, matrix_d, height*width*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(matrix_d);

    printf("Saving GPU output... ");
    // save changes in binary data matrix from GPU
	for(int i = 0; i < height; i++) {
		for(int j = 0; j < width*3; j += 3) {
			int im = height-i-1; 
			int jm = j/3;
            rows[i][j] = matrix_out_d[im*width + jm];
			rows[i][j+1] = matrix_out_d[im*width + jm];
			rows[i][j+2] = matrix_out_d[im*width + jm];
        }
    }
	// Write GPU output
    saveBmp(info,rows,row_padded,height,"GPU_out_%d.bmp");
    printf(" => Done \n");
    fclose(fr);

    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
	double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;
    printf("\n******************************** FINAL OUT PUT ********************************\n");
	printf("Image dimensions: (%d,%d) \nGPU Time: %f \nCPU Time: %f\n",width,height,time_d,time_h);
    printf("*********************************************************************************\n");
    return 0;
}

